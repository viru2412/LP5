
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

#define checkCudaErrors(call)                                                                 \
    do {                                                                                      \
        hipError_t err = call;                                                               \
        if (err != hipSuccess) {                                                             \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                               \
        }                                                                                     \
    } while (0)

using namespace std;

// VectorAdd parallel function
__global__ void vectorAdd(int *a, int *b, int *result, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        result[tid] = a[tid] + b[tid];
    }
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 1 << 6;

    a = new int[n];
    b = new int[n];
    c = new int[n];
    int *d = new int[n];
    int size = n * sizeof(int);
    checkCudaErrors(hipMalloc(&a_dev, size));
    checkCudaErrors(hipMalloc(&b_dev, size));
    checkCudaErrors(hipMalloc(&c_dev, size));

    // Array initialization..You can use Randon function to assign values
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 10000;
        b[i] = rand() % 10000;
        d[i] = a[i] + b[i];  // calculating serial addition
    }
    cout << "Given array A is =>\n";
    for (int i = 0; i < n; i++) {
        cout << a[i] << ", ";
    }
    cout << "\n\n";

    cout << "Given array B is =>\n";
    for (int i = 0; i < n; i++) {
        cout << b[i] << ", ";
    }
    cout << "\n\n";

    hipEvent_t start, end;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    checkCudaErrors(hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice));
    int threads = 1024;
    int blocks = (n + threads - 1) / threads;
    checkCudaErrors(hipEventRecord(start));

    // Parallel addition program
    vectorAdd<<<blocks, threads>>>(a_dev, b_dev, c_dev, n);

    checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipEventSynchronize(end));

    float time = 0.0;
    checkCudaErrors(hipEventElapsedTime(&time, start, end));

    checkCudaErrors(hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost));

    // Calculate the error term.

    cout << "CPU sum is =>\n";
    for (int i = 0; i < n; i++) {
        cout << d[i] << ", ";
    }
    cout << "\n\n";

    cout << "GPU sum is =>\n";
    for (int i = 0; i < n; i++) {
        cout << c[i] << ", ";
    }
    cout << "\n\n";

    int error = 0;
    for (int i = 0; i < n; i++) {
        error += d[i] - c[i];
        if (0 != (d[i] - c[i])) {
            cout << "Error at (" << i << ") => GPU: " << c[i] << ", CPU: " << d[i] << "\n";
        }
    }

    cout << "\nError : " << error;
    cout << "\nTime Elapsed: " << time;

    return 0;
}

/*

OUTPUT:

Given array A is =>
383, 777, 793, 386, 649, 362, 690, 763, 540, 172, 211, 567, 782, 862, 67, 929,

Given array B is =>
886, 915, 335, 492, 421, 27, 59, 926, 426, 736, 368, 429, 530, 123, 135, 802,

CPU sum is =>
1269, 1692, 1128, 878, 1070, 389, 749, 1689, 966, 908, 579, 996, 1312, 985, 202, 1731,

GPU sum is =>
1269, 1692, 1128, 878, 1070, 389, 749, 1689, 966, 908, 579, 996, 1312, 985, 202, 1731,


Error : 0
Time Elapsed:  0.017408

*/
