
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <iostream>

#define checkCudaErrors(call)                                                                 \
    do {                                                                                      \
        hipError_t err = call;                                                               \
        if (err != hipSuccess) {                                                             \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                               \
        }                                                                                     \
    } while (0)

using namespace std;

// Matrix multiplication Cuda
__global__ void matrixMultiplication(int *a, int *b, int *c, int n) {
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;

    if (row < n && col < n)
        for (int j = 0; j < n; j++) {
            sum = sum + a[row * n + j] * b[j * n + col];
        }

    c[n * row + col] = sum;
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 10;

    a = new int[n * n];
    b = new int[n * n];
    c = new int[n * n];
    int *d = new int[n * n];
    int size = n * n * sizeof(int);
    checkCudaErrors(hipMalloc(&a_dev, size));
    checkCudaErrors(hipMalloc(&b_dev, size));
    checkCudaErrors(hipMalloc(&c_dev, size));

    // Array initialization
    for (int i = 0; i < n * n; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    cout << "Given matrix A is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << a[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    cout << "Given matrix B is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << b[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    hipEvent_t start, end;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    checkCudaErrors(hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);

    // GPU Multiplication
    checkCudaErrors(hipEventRecord(start));
    matrixMultiplication<<<blocksPerGrid, threadsPerBlock>>>(a_dev, b_dev, c_dev, n);

    checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipEventSynchronize(end));

    float time = 0.0;
    checkCudaErrors(hipEventElapsedTime(&time, start, end));

    checkCudaErrors(hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost));

    // CPU matrix multiplication
    int sum = 0;
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            sum = 0;
            for (int k = 0; k < n; k++) sum = sum + a[row * n + k] * b[k * n + col];
            d[row * n + col] = sum;
        }
    }

    cout << "CPU product is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << d[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    cout << "GPU product is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << c[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    int error = 0;
    int _c, _d;
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            _c = c[row * n + col];
            _d = d[row * n + col];
            error += _c - _d;
            if (0 != (_c - _d)) {
                cout << "Error at (" << row << ", " << col << ") => GPU: " << _c << ", CPU: " << _d
                     << "\n";
            }
        }
    }
    cout << "\n";

    cout << "Error : " << error;
    cout << "\nTime Elapsed: " << time;

    return 0;
}

/*

OUTPUT:

Given matrix A is =>
3 7 3 6 9 2 0 3 0 2
1 7 2 2 7 9 2 9 3 1
9 1 4 8 5 3 1 6 2 6
5 4 6 6 3 4 2 4 4 3
7 6 8 3 4 2 6 9 6 4
5 4 7 7 7 2 1 6 5 4
0 1 7 1 9 7 7 6 6 9
8 2 3 0 8 0 6 8 6 1
9 4 1 3 4 4 7 3 7 9
2 7 5 4 8 9 5 8 3 8

Given matrix B is =>
6 5 5 2 1 7 9 6 6 6
8 9 0 3 5 2 8 7 6 2
3 9 7 4 0 6 0 3 0 1
5 7 5 9 7 5 5 7 4 0
8 8 4 1 9 0 8 2 6 9
0 8 1 2 2 6 0 1 9 9
9 7 1 5 7 6 3 5 3 4
1 9 9 8 5 9 3 5 1 5
8 8 0 0 4 4 6 1 5 6
1 8 7 1 5 7 3 8 1 9

CPU product is =>
190 278 145 132 190 136 200 169 161 167
186 355 156 157 207 209 185 164 210 246
191 335 233 179 196 257 220 227 174 232
191 319 172 156 167 218 182 186 165 186
276 433 239 205 229 305 251 252 193 257
233 378 222 181 218 240 231 216 180 226
232 430 221 155 255 274 187 203 193 328
248 319 178 137 201 217 233 171 165 236
267 379 184 141 231 276 259 247 218 301
252 477 239 204 282 302 239 261 245 334

GPU product is =>
190 278 145 132 190 136 200 169 161 167
186 355 156 157 207 209 185 164 210 246
191 335 233 179 196 257 220 227 174 232
191 319 172 156 167 218 182 186 165 186
276 433 239 205 229 305 251 252 193 257
233 378 222 181 218 240 231 216 180 226
232 430 221 155 255 274 187 203 193 328
248 319 178 137 201 217 233 171 165 236
267 379 184 141 231 276 259 247 218 301
252 477 239 204 282 302 239 261 245 334


Error : 0
Time Elapsed: 0.018144

*/
